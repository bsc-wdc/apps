#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "kernel.h"

static inline int nextPowerOfTwo(int n) {
    n--;

    n = n >>  1 | n;
    n = n >>  2 | n;
    n = n >>  4 | n;
    n = n >>  8 | n;
    n = n >> 16 | n;
//  n = n >> 32 | n;    //  For 64-bit ints

    return ++n;
}

/*----< euclid_dist_2() >----------------------------------------------------*/
/* square of Euclid distance between two multi-dimensional points            */
 __host__ __device__ inline static
float euclid_dist_2(int    numCoords,
                    int    numObjs,
                    int    numClusters,
                    float *objects,     // [numCoords][numObjs]
                    float *clusters,    // [numCoords][numClusters]
                    int    objectId,
                    int    clusterId)
{   
    int i;
    float ans=0.0;

    if(objectId == 0){
    	printf("coords used in first element\n");
    	for (i = 0; i < numCoords; i++){
		printf("%f\n", objects[numObjs * i + objectId]);
    	}
	printf("cluster coords\n");
	for (i = 0; i < numCoords; i++){
                printf("%f\n", clusters[numClusters * i + clusterId]);
        }
    }

                                                                                                     
    for (i = 0; i < numCoords; i++) { 
        ans += (objects[numObjs * i + objectId] - clusters[numClusters * i + clusterId]) *
               (objects[numObjs * i + objectId] - clusters[numClusters * i + clusterId]);
    }
     
    if(objectId == 0){
	printf("first ans is %f for cluster %d\n", ans, clusterId);
    }
                                                                                                                                                                        
    return(ans);
}

/*__global__ void cuda_test(int n, int *objects, int *out){

}
*/

/*
__global__
void cuda_find_nearest_cluster(int numCoords,
                          int numObjs,
                          int numClusters,
                          float *objects,           //  [numCoords][numObjs]
                          float *deviceClusters,    //  [numCoords][numClusters]
                          int *membership,
                          int numThreadsPerClusterBlock,
                          int numClusterBlocks,
                          int clusterBlockSharedDataSize)
{*/






/*----< cuda_find_nearest_cluster() >---------------------------------------------*/

__global__ 
void cuda_find_nearest_cluster(int numCoords,
                          int numObjs,
                          int numClusters,
			  float *objects,           //  [numCoords][numObjs]
                          float *deviceClusters,    //  [numCoords][numClusters]
                          int *membership,
			  int numThreadsPerClusterBlock, 
			  int numClusterBlocks, 
			  int clusterBlockSharedDataSize
			)
{
  
//   printf("hello from cuda task \n"); 



   //extern __shared__ char sharedMemory[];

    __shared__ char sharedMemory[128];

    unsigned char *membershipChanged = (unsigned char *)sharedMemory;

//#if BLOCK_SHARED_MEM_OPTIMIZATION
//    float *clusters = (float *)(sharedMemory + blockDim.x);
//#else
    float *clusters = deviceClusters;
//#endif
    
    membershipChanged[threadIdx.x] = 0;

//#if BLOCK_SHARED_MEM_OPTIMIZATION
    //  BEWARE: We can overrun our shared memory here if there are too many
    //  clusters or too many coordinates! For reference, a Tesla C1060 has 16
    //  KiB of shared memory per block, and a GeForce GTX 480 has 48 KiB of
    //  shared memory per block.
//    for (int i = threadIdx.x; i < numClusters; i += blockDim.x) {
//        for (int j = 0; j < numCoords; j++) {
//    	    clusters[numClusters * j + i] = deviceClusters[numClusters * j + i];
//        }
//    }
//    __syncthreads();
//#endif
    
    int objectId = blockDim.x * blockIdx.x + threadIdx.x;


    if (objectId < numObjs) {
//	printf("task %d reporting, nObjects is %d, values are %d \n %d \n %d \n %d \n %d \n %d \n %d \n %d \n %d \n %d \n\n", objectId, numObjs, objects[objectId], objects[numObjs * 1 + objectId], objects[numObjs * 2 + objectId], objects[numObjs * 3 + objectId], objects[numObjs * 4 + objectId], objects[numObjs * 5 + objectId], objects[numObjs * 6 + objectId], objects[numObjs * 7 + objectId], objects[numObjs * 8 + objectId], objects[numObjs * 9 + objectId]);


/*
	if (objectId == 0){
		printf("objects:\n");
		for (int i = 0; i < numObjs*numCoords; i++){
			printf("%f\n", objects[i]);
		}
	}
*/




//	printf("task %d reporting, membership values are %d \n %d \n %d \n %d \n %d \n %d \n %d \n %d \n %d \n %d \n\n", objectId, membership[0], membership[1], membership[2], membership[3], membership[4], membership[5], membership[6], membership[7], membership[8], membership[9]);	

        int   index, i;
        float dist, min_dist;

	// find the cluster id that has min distance to object 
        index    = 0;
        min_dist = euclid_dist_2(numCoords, numObjs, numClusters,
                                 objects, clusters, objectId, 0);
	__syncthreads();
	printf("distance 0 for object is %f\n", objectId, min_dist);
	
        for (i=1; i<numClusters; i++) {
            dist = euclid_dist_2(numCoords, numObjs, numClusters,
                                 objects, clusters, objectId, i);
            // no need square root 
		printf("distance 1 for object %d is %f\n", objectId, dist);
            if (dist < min_dist) { // find the min and its array index 
                min_dist = dist;
                index    = i;
            }
        }
	__syncthreads();
	printf("chosen dist for object %d is %f\n", objectId, min_dist);


	if (membership[objectId] != index) {
            membershipChanged[threadIdx.x] = 1;
        }

//	printf("object id %d has membership %d\n", objectId, index);	

        // assign the membership to object objectId 
        membership[objectId] = index;

        __syncthreads();    //  For membershipChanged[]

	printf("object id %d has membership %d\n", objectId, index);

	for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
            if (threadIdx.x < s) {
                membershipChanged[threadIdx.x] +=
                    membershipChanged[threadIdx.x + s];
            }
            __syncthreads();
        }

//	membership[objectId] = 123;

    }

/*
    int objectId = blockDim.x * blockIdx.x + threadIdx.x;


    if (objectId < numObjs) {
	membership[objectId] = 123;
    }
*/

	printf("end of cuda task \n");

}
